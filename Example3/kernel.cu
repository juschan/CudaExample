#include "hip/hip_runtime.h"
//Example 3.2.2
#include <stdio.h>
#include "../common/book.h"

__global__ void multiply(int a, int b, int *c) {
	*c = a * b;
}

int main(void) {
	int c;
	int *dev_c;

	HANDLE_ERROR(hipMalloc((void**)&dev_c, sizeof(int)));

	multiply <<< 1, 1 >>> (4, 11, dev_c);

	HANDLE_ERROR(hipMemcpy(&c, dev_c, sizeof(int), hipMemcpyDeviceToHost));

	printf("4 * 11 = %d\n", c);

	hipFree(dev_c);
	return 0;
}