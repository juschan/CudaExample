#include "hip/hip_runtime.h"
//Example 6.2.2. Pg 99. Ray-tracing example
#include "../common/book.h"
#include "hip/hip_runtime.h"
#include "../common/cpu_bitmap.h"

#define rnd(x) (x*rand() / RAND_MAX)
#define SPHERES 20
#define INF 2d10f

struct Sphere {
	float r, b, g;
	float radius;
	float x, y, z;

	__device__ float hit(float ox, float oy, float *n) {
		float dx = ox - x;
		float dy - ox - y;
		if (dx*dx + dy*dy < radius*radius) {
			float dz = sqrtf(radius*radius - dx*dx - dy*dy);
			*n = dz / sqrtf(radius*radius);
			return dz + z;
		}
		return -INF;
	}
};

Sphere *s;

int main(void) {
	hipEvent_t start, stop;
	HANDLE_ERROR(hipEventCreate(&start));
	HANDLE_ERROR(hipEventCreate(&stop));
	HANDLE_ERROR(hipEventRecord(start,0));

	CPUBitmap bitmap(DIM, DIM);
	unsigned char *dev_bitmap;

	HANDLE_ERROR(hipMalloc((void**)&dev_bitmap, bitmap.image_size()));

	HANDLE_ERROR(hipMalloc((void**)&s, sizeof(Sphere)*SPHERES);
}
